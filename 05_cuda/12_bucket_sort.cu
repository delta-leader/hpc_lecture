
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucket_sort(int *key, int* bucket){
  atomicAdd(&bucket[key[threadIdx.x]], 1);
  __syncthreads();
  int buck_val = 0;
  for (int i = threadIdx.x; i >= bucket[buck_val]; i-=bucket[buck_val++]);
  key[threadIdx.x] = buck_val;
}

int main() {
  int n = 50;
  int range = 5;
  int *key, *bucket;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  // initialize to 0
  hipMemset(bucket, 0, range);

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  //only launching 1 block because n is small, could be extended but would need cooperative_groups for grid synchronization
  bucket_sort<<<1, n>>>(key, bucket);
  hipDeviceSynchronize();
  

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  /*for (int i=0; i<range; i++) {
    printf("%d ",bucket[i]);
  }
  printf("\n");*/
  
  hipFree(key);
  hipFree(bucket);
}
